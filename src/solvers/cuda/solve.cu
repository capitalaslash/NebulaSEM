#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "solve.h"

/*number of threads in a block*/
static const Int nThreads = 128;

/*Matrix vector multiply*/
template <class T>
__global__
void cudaMul(const Int* const rows,
             const Int* const cols,
             const Scalar* const an,
             const Int N,
             const T* const x, 
             T* y
             ) {
    Int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)  {
        const Int start = rows[i];
        const Int end = rows[i + 1];
        T res = an[start] * x[cols[start]];

        for (Int j = start + 1; j < end; j++)
            res -= an[j] * x[cols[j]];
        y[i] = res;
    }
}
/*jacobi solver*/
template<class T>
__global__
void cudaJacobi(const Int* const rows,
                 const Int* const cols,
                 const Scalar* const an,
                 const T* const cF,
                 T* const cF1,
                 const T* const Su,
                 T* r,
                 const Int N, 
                 Scalar omega
                 ) {
    Int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)  {
        const Int start = rows[i];
        const Int end = rows[i + 1];
        T res = Su[i], val = cF[i];

        for (Int j = start + 1; j < end; j++)
            res += an[j] * cF[cols[j]];
        res /= an[start];

        r[i] = -val;
        val *= (1 - omega);
        val += res * (omega);
        r[i] += val;
        cF1[i] = val;
    }
}
/*Taxpy*/
template<class T,class T1>
__global__
void cudaTaxpy(const Int N,
               const T1 alpha,
               const T* const x,
               const T* const y,
               T* const z
               ) {
    Int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)  {
        T temp;
        temp = x[i];
        temp *= alpha;
        temp += y[i];
        z[i] = temp;
    }
}
/*Txmy*/
template<class T,class T1>
__global__
void cudaTxmy(const Int N,
              const T* const x,
              const T1* const y,
              T* const z
              ) {
    Int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)  {
        T temp;
        temp = x[i];
        temp *= y[i];
        z[i] = temp;
    }
}
/*Tdot*/
template <class T>
__global__ 
void Tdot(const T* const a, 
          const T* const b, 
          T* const c, 
          const Int N
          ) {
    __shared__ T cache[nThreads];
    Int tid = threadIdx.x + blockIdx.x * blockDim.x;
    Int cacheIndex = threadIdx.x;

    T   temp = T(0),val;
    while (tid < N) {
        val = a[tid];
        val *= b[tid];
        temp += val;
        tid += blockDim.x * gridDim.x;
    }  
    cache[cacheIndex] = temp;
    
    __syncthreads();

    Int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}
template<class T>
__host__ 
T cudaTdot(T* x,
           T* y,
           T* d_sum,
           T* sum,
           const Int nBlocks32,
           const Int N
           ) {
    Tdot <<< nBlocks32, nThreads >>> (x,y,d_sum,N);
    hipMemcpy(sum,d_sum,nBlocks32 * sizeof(T),hipMemcpyDeviceToHost);
    T c = T(0);
    for (Int i = 0; i < nBlocks32; i++)
        c += sum[i];
    return c;
}
/* **************************************
 *   CSR - compressed sparse row format
 *       * Used for on GPU computation
 *       * Propably for AMG too
 * **************************************/
template <class T>
class CSRMatrix {
public:
    std::vector<Int>  rows;
    std::vector<Int>  cols;
    std::vector<Scalar> an;
    std::vector<Scalar> anT;
    std::vector<T> cF;
    std::vector<T> Su;
public:
    template <class T1>
    CSRMatrix(const MeshMatrix<T1>& A) {
        using namespace Mesh;
        using namespace DG;
        const Int N  = A.ap.size();
        const Int NN = A.ap.size() + 
                       A.an[0].size() + 
                       A.an[1].size() +
                       (NPMAT ? (A.adg.size() - A.ap.size()) : 0); 
        register Int i,f;

        /*resize*/
        cF.resize(N);
        Su.resize(N);
        rows.reserve(N + 1);
        cols.reserve(NN);
        an.reserve(NN);
        anT.reserve(NN);

        /*source term*/
        for(i = 0;i < N;i++) {
            Su[i] = A.Su[i];
            cF[i] = (*A.cF)[i];
        }

        /*fill matrix in CSR format.Diagonal element 
          is always at the start of a row */
        Int cn = 0;
        for(ii = 0;ii < gCells.size();ii++) {
            Cell& c = gCells[ii];
            for(Int j = 0;j < NP;j++) {
                Int i = ii * NP + j;

                rows.push_back(cn);

                an.push_back(A.ap[i]);
                anT.push_back(A.ap[i]);
                cols.push_back(i);
                cn++;

                forEach(c,k) {
                    f = c[k];
                    if(i == gFO[f]) {
                        an.push_back(A.an[1][f]);
                        anT.push_back(A.an[0][f]);
                        cols.push_back(gFN[f]);
                        cn++;
                    } else {
                        an.push_back(A.an[0][f]);
                        anT.push_back(A.an[1][f]);
                        cols.push_back(gFO[f]);
                        cn++;
                    }
                }
                
                if(NPMAT) {
                    for(Int k = 0;k < NP;k++) {
                        if(k == j) continue;
                        an.push_back(A.adg[ii * NPMAT + j * NP + k]);
                        anT.push_back(A.adg[ii * NPMAT + k * NP + j]);
                        cols.push_back();
                        cn++;
                    }
                }
            }
        }
        /*push extra row*/
        rows.push_back(cn);
    }
    /*IO*/
    template<typename Ts>
    friend Ts& operator << (Ts& os, const CSRMatrix& p) {
        os << p.rows << "\n";
        os << p.cols << "\n";
        os << p.an << "\n";
        os << p.Su << "\n";
        return os;
    }
    template<typename Ts>
    friend Ts& operator >> (Ts& is, CSRMatrix& p) {
        is >> p.rows;
        is >> p.cols;
        is >> p.an;
        is >> p.Su;
        return is;
    }
    /*end*/
};
/***********************************************
 * Template class to solve equations on GPU
 *      Solver must do many iterations to compensate
 *      for the latency caused by copying matrix
 *      from host to device.
 ***********************************************/
template<class T>
__host__
void SolveT(const MeshMatrix<T>& M) {
    const Int N = Mesh::gBCSfield;
    const Int Nall = M.ap.size();
    const Int nBlocks = (N + nThreads - 1) / nThreads;
    const Int nBlocks32 = ((nBlocks > 32) ? 32 : nBlocks);

    //info
    if(M.flags & M.SYMMETRIC)
        MP::printH("Symmetric  : ");
    else
        MP::printH("Asymmetric : ");
    if(Controls::Solver == Controls::SOR)
        MP::print("SOR :");
    else
        MP::print("PCG :");

    /*******************************
     *  variables on host & device
     *******************************/
    Int*   d_rows;
    Int*   d_cols;
    Scalar*  d_an;
    Scalar*  d_anT;
    Scalar*  d_pC;
    T*       d_cF;
    T*       d_Su;
    //PCG
    T*       d_r,*d_r1;
    T*       d_p,*d_p1,*d_AP,*d_AP1;
    T        alpha,beta,o_rr,oo_rr;
    T        local_res[2];
    //reduction
    T*       sum,*d_sum;

    /*********************************
     * allocate memory on device
     ********************************/
    {
        CSRMatrix<T> A(M);  
        hipMalloc((void**) &d_rows,A.rows.size() * sizeof(Int));
        hipMalloc((void**) &d_cols,A.cols.size() * sizeof(Int));
        hipMalloc((void**) &d_an,  A.an.size() * sizeof(Scalar));
        hipMalloc((void**) &d_cF,  Nall * sizeof(T));
        hipMalloc((void**) &d_Su,  Nall * sizeof(T));

        hipMemcpy(d_rows ,&A.rows[0] ,A.rows.size() * sizeof(Int),  hipMemcpyHostToDevice);
        hipMemcpy(d_cols ,&A.cols[0] ,A.cols.size() * sizeof(Int),  hipMemcpyHostToDevice);
        hipMemcpy(d_an   ,&A.an[0]   ,A.an.size() * sizeof(Scalar), hipMemcpyHostToDevice);
        hipMemcpy(d_cF   ,&A.cF[0]   ,Nall *   sizeof(T),    hipMemcpyHostToDevice);
        hipMemcpy(d_Su   ,&A.Su[0]   ,Nall *   sizeof(T),    hipMemcpyHostToDevice);

        hipMalloc((void**) &d_r, Nall * sizeof(T));
        hipMalloc((void**) &d_sum, nBlocks32 * sizeof(T));
        sum = (T*) malloc(nBlocks32 * sizeof(T));

        if(Controls::Solver == Controls::SOR) {
            hipMalloc((void**) &d_AP,Nall * sizeof(T));
            hipMemcpy( d_AP,d_cF,Nall * sizeof(T),hipMemcpyDeviceToDevice);
        } else if(Controls::Solver == Controls::PCG) {
            hipMalloc((void**) &d_p,   Nall * sizeof(T));
            hipMalloc((void**) &d_AP,  Nall * sizeof(T));
            {
                ScalarCellField pC = 1./M.ap;
                hipMalloc((void**) &d_pC,N * sizeof(Scalar));
                hipMemcpy(d_pC,&pC[0],N * sizeof(Scalar),hipMemcpyHostToDevice);
            }
            if(!(M.flags & M.SYMMETRIC)) {
                hipMalloc((void**) &d_r1,   Nall * sizeof(T));
                hipMalloc((void**) &d_p1,   Nall * sizeof(T));
                hipMalloc((void**) &d_AP1,  Nall * sizeof(T));
                hipMalloc((void**) &d_anT,A.anT.size() * sizeof(Scalar));
                hipMemcpy(d_anT,&A.anT[0],A.anT.size() * sizeof(Scalar), hipMemcpyHostToDevice);
            }
        }
    }

    /*CG*/
    if(Controls::Solver == Controls::PCG) {
        hipMemset(d_r,0,Nall * sizeof(T));
        hipMemset(d_p,0,Nall * sizeof(T));
        cudaMul   <<< nBlocks, nThreads >>> (d_rows,d_cols,d_an,N,d_cF,d_AP);
        cudaTaxpy <<< nBlocks, nThreads >>> (N,Scalar(-1),d_AP,d_Su,d_r);
        cudaTxmy  <<< nBlocks, nThreads >>> (N,d_r,d_pC,d_p);
        o_rr = cudaTdot(d_r,d_p,d_sum,sum,nBlocks32,N);
    }
    /*BiCG*/
    if(!(M.flags & M.SYMMETRIC) && (Controls::Solver == Controls::PCG)) {
        hipMemcpy(d_r1,d_r,Nall * sizeof(T), hipMemcpyDeviceToDevice);
        hipMemcpy(d_p1,d_p,Nall * sizeof(T), hipMemcpyDeviceToDevice);
    }
    //iterate until convergence
    Scalar res = 0;
    Int iterations = 0;

    /* **************************
     * Iterative solvers
     * *************************/
    while(iterations < Controls::max_iterations) {
        /*counter*/
        iterations++;

        /*select solver*/
        if(Controls::Solver == Controls::SOR) {
            iterations++;
            cudaJacobi <<< nBlocks, nThreads >>> (d_rows,d_cols,d_an,d_cF,d_AP,d_Su,d_r,N,Controls::SOR_omega);
            cudaJacobi <<< nBlocks, nThreads >>> (d_rows,d_cols,d_an,d_AP,d_cF,d_Su,d_r,N,Controls::SOR_omega);
        } else if(M.flags & M.SYMMETRIC) {
            /*conjugate gradient   : from wiki*/
            cudaMul <<< nBlocks, nThreads >>> (d_rows,d_cols,d_an,N,d_p,d_AP);
            oo_rr = cudaTdot(d_p,d_AP,d_sum,sum,nBlocks32,N);
            alpha = sdiv(o_rr , oo_rr);
            cudaTaxpy <<< nBlocks, nThreads >>> (N,alpha,d_p,d_cF,d_cF);
            cudaTaxpy <<< nBlocks, nThreads >>> (N,-alpha,d_AP,d_r,d_r);
            oo_rr = o_rr;
            cudaTxmy <<< nBlocks, nThreads >>> (N,d_r,d_pC,d_AP);
            o_rr = cudaTdot(d_r,d_AP,d_sum,sum,nBlocks32,N);
            beta = sdiv(o_rr , oo_rr);
            cudaTaxpy <<< nBlocks, nThreads >>> (N,beta,d_p,d_AP,d_p);
            /*end*/
        } else {
            /* biconjugate gradient : from wiki */
            cudaMul <<< nBlocks, nThreads >>> (d_rows,d_cols,d_an,N,d_p,d_AP);
            cudaMul <<< nBlocks, nThreads >>> (d_rows,d_cols,d_anT,N,d_p1,d_AP1);
            oo_rr = cudaTdot(d_p1,d_AP,d_sum,sum,nBlocks32,N);
            alpha = sdiv(o_rr , oo_rr);
            cudaTaxpy <<< nBlocks, nThreads >>> (N,alpha,d_p,d_cF,d_cF);
            cudaTaxpy <<< nBlocks, nThreads >>> (N,-alpha,d_AP,d_r,d_r);
            cudaTaxpy <<< nBlocks, nThreads >>> (N,-alpha,d_AP1,d_r1,d_r1);
            oo_rr = o_rr;
            cudaTxmy <<< nBlocks, nThreads >>> (N,d_r,d_pC,d_AP);
            cudaTxmy <<< nBlocks, nThreads >>> (N,d_r1,d_pC,d_AP1);
            o_rr = cudaTdot(d_r1,d_AP,d_sum,sum,nBlocks32,N);
            beta = sdiv(o_rr , oo_rr);
            cudaTaxpy <<< nBlocks, nThreads >>> (N,beta,d_p,d_AP,d_p);
            cudaTaxpy <<< nBlocks, nThreads >>> (N,beta,d_p1,d_AP1,d_p1);
        }

        /* *********************************************
        * calculate norm of residual & check convergence
        * **********************************************/
        local_res[0] = cudaTdot(d_r,d_r,d_sum,sum,nBlocks32,N);
        local_res[1] = cudaTdot(d_cF,d_cF,d_sum,sum,nBlocks32,N);
        res = sqrt(mag(local_res[0]) / mag(local_res[1]));
        
        /*check convergence*/
        if(res <= Controls::tolerance)
            break;
    }

    /*****************************
     *  Copy result back to cpu
     *****************************/
    //copy result
    hipMemcpy(&((*M.cF)[0]), d_cF, N * sizeof(T), hipMemcpyDeviceToHost);

    //update boundary conditons
    updateExplicitBCs(*M.cF);

    //info
    MP::print("Iterations %d Residue: %.5e\n",iterations,res);
    /*********************************
     * free device memory
     ********************************/
    {
        hipFree(d_rows);
        hipFree(d_cols);
        hipFree(d_an);
        hipFree(d_cF);
        hipFree(d_Su);

        hipFree(d_r);
        hipFree(d_sum);
        free(sum);

        if(Controls::Solver == Controls::SOR) {
            hipFree(d_AP);
        } else if(Controls::Solver == Controls::PCG) {
            hipFree(d_p);
            hipFree(d_AP);
            hipFree(d_pC);
            if(!(M.flags & M.SYMMETRIC)) {
                hipFree(d_r1);
                hipFree(d_p1);
                hipFree(d_AP1);
                hipFree(d_anT);
            }
        }
    }
    /******************
     *    END
     ******************/
}

/***************************
 * Explicit instantiations
 ***************************
void Solve(const MeshMatrix<Scalar>& A) {
    applyImplicitBCs(A);
    SolveT(A);
}
void Solve(const MeshMatrix<Vector>& A) {
    applyImplicitBCs(A);
    SolveT(A);
}
void Solve(const MeshMatrix<STensor>& A) {
    applyImplicitBCs(A);
    SolveT(A);
}
void Solve(const MeshMatrix<Tensor>& A) {
    applyImplicitBCs(A);
    SolveT(A);
}
/* ********************
 *        End
 * ********************/
